#include "npp.h"
#include <cstdint>
#include <hip/hip_runtime.h>


// Generic multi-channel copy kernel template
template <typename T, int CHANNELS>
__global__ void nppiCopy_kernel(const T *pSrc, int nSrcStep, T *pDst, int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const T *src_row = (const T *)((const char *)pSrc + y * nSrcStep);
  T *dst_row = (T *)((char *)pDst + y * nDstStep);

#pragma unroll
  for (int c = 0; c < CHANNELS; c++) {
    dst_row[x * CHANNELS + c] = src_row[x * CHANNELS + c];
  }
}

// Optimized vectorized kernel for 8u_C4
__global__ void nppiCopy_8u_C4R_vectorized_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                                  int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const uint32_t *src_row = (const uint32_t *)((const char *)pSrc + y * nSrcStep);
  uint32_t *dst_row = (uint32_t *)((char *)pDst + y * nDstStep);

  dst_row[x] = src_row[x];
}

// Packed to planar copy kernel for 32f_C3P3R
__global__ void nppiCopy_32f_C3P3R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *const *pDst, int nDstStep,
                                          int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp32f *src_row = (const Npp32f *)((const char *)pSrc + y * nSrcStep);

#pragma unroll
  for (int c = 0; c < 3; c++) {
    Npp32f *dst_row = (Npp32f *)((char *)pDst[c] + y * nDstStep);
    dst_row[x] = src_row[x * 3 + c];
  }
}

// Generic copy implementation template (outside extern "C")
template <typename T, int CHANNELS>
NppStatus nppiCopy_impl(const T *pSrc, int nSrcStep, T *pDst, int nDstStep, NppiSize oSizeROI,
                        NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_kernel<T, CHANNELS><<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                 oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

extern "C" {

// Explicit instantiations for 8u
NppStatus nppiCopy_8u_C1R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp8u, 1>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_8u_C3R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp8u, 3>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_8u_C4R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  if (oSizeROI.width % 4 == 0 && nSrcStep % 4 == 0 && nDstStep % 4 == 0) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);
    nppiCopy_8u_C4R_vectorized_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pDst, nDstStep, oSizeROI.width, oSizeROI.height);
  } else {
    return nppiCopy_impl<Npp8u, 4>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
  }

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }
  return NPP_SUCCESS;
}

// Explicit instantiations for 16u
NppStatus nppiCopy_16u_C1R_Ctx_impl(const Npp16u *pSrc, int nSrcStep, Npp16u *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16u, 1>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_16u_C3R_Ctx_impl(const Npp16u *pSrc, int nSrcStep, Npp16u *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16u, 3>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_16u_C4R_Ctx_impl(const Npp16u *pSrc, int nSrcStep, Npp16u *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16u, 4>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

// Explicit instantiations for 16s
NppStatus nppiCopy_16s_C1R_Ctx_impl(const Npp16s *pSrc, int nSrcStep, Npp16s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16s, 1>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_16s_C3R_Ctx_impl(const Npp16s *pSrc, int nSrcStep, Npp16s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16s, 3>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_16s_C4R_Ctx_impl(const Npp16s *pSrc, int nSrcStep, Npp16s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp16s, 4>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

// Explicit instantiations for 32s
NppStatus nppiCopy_32s_C1R_Ctx_impl(const Npp32s *pSrc, int nSrcStep, Npp32s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32s, 1>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_32s_C3R_Ctx_impl(const Npp32s *pSrc, int nSrcStep, Npp32s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32s, 3>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_32s_C4R_Ctx_impl(const Npp32s *pSrc, int nSrcStep, Npp32s *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32s, 4>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

// Explicit instantiations for 32f
NppStatus nppiCopy_32f_C1R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32f, 1>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_32f_C3R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32f, 3>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

NppStatus nppiCopy_32f_C4R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  return nppiCopy_impl<Npp32f, 4>(pSrc, nSrcStep, pDst, nDstStep, oSizeROI, nppStreamCtx);
}

// Packed to planar copy implementation
NppStatus nppiCopy_32f_C3P3R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *const pDst[3], int nDstStep,
                                      NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  Npp32f **d_pDst;

  hipError_t cudaStatus = hipMalloc(&d_pDst, 3 * sizeof(Npp32f *));
  if (cudaStatus != hipSuccess) {
    return NPP_MEMORY_ALLOCATION_ERR;
  }

  cudaStatus = hipMemcpy(d_pDst, pDst, 3 * sizeof(Npp32f *), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    hipFree(d_pDst);
    return NPP_MEMORY_ALLOCATION_ERR;
  }

  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_32f_C3P3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, d_pDst, nDstStep,
                                                                              oSizeROI.width, oSizeROI.height);

  cudaStatus = hipGetLastError();
  hipFree(d_pDst);

  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}
}
