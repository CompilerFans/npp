#include "npp.h"
#include <cstdint>
#include <hip/hip_runtime.h>


/**
 * CUDA kernels for NPP Image Copy Functions
 * Implements efficient GPU-based image copying operations
 */

// Kernel for 8-bit unsigned single channel copy
__global__ void nppiCopy_8u_C1R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *src_row = (const Npp8u *)((const char *)pSrc + y * nSrcStep);
  Npp8u *dst_row = (Npp8u *)((char *)pDst + y * nDstStep);

  dst_row[x] = src_row[x];
}

// Kernel for 8-bit unsigned three channel copy
__global__ void nppiCopy_8u_C3R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *src_row = (const Npp8u *)((const char *)pSrc + y * nSrcStep);
  Npp8u *dst_row = (Npp8u *)((char *)pDst + y * nDstStep);

  // Copy all three channels
  dst_row[x * 3 + 0] = src_row[x * 3 + 0]; // R or B
  dst_row[x * 3 + 1] = src_row[x * 3 + 1]; // G
  dst_row[x * 3 + 2] = src_row[x * 3 + 2]; // B or R
}

// Kernel for 8-bit unsigned four channel copy
__global__ void nppiCopy_8u_C4R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *src_row = (const Npp8u *)((const char *)pSrc + y * nSrcStep);
  Npp8u *dst_row = (Npp8u *)((char *)pDst + y * nDstStep);

  // Copy all four channels (RGBA/BGRA)
  dst_row[x * 4 + 0] = src_row[x * 4 + 0];
  dst_row[x * 4 + 1] = src_row[x * 4 + 1];
  dst_row[x * 4 + 2] = src_row[x * 4 + 2];
  dst_row[x * 4 + 3] = src_row[x * 4 + 3];
}

// Kernel for 32-bit float single channel copy
__global__ void nppiCopy_32f_C1R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, int width,
                                        int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp32f *src_row = (const Npp32f *)((const char *)pSrc + y * nSrcStep);
  Npp32f *dst_row = (Npp32f *)((char *)pDst + y * nDstStep);

  dst_row[x] = src_row[x];
}

// Kernel for 32-bit float three channel copy
__global__ void nppiCopy_32f_C3R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, int width,
                                        int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp32f *src_row = (const Npp32f *)((const char *)pSrc + y * nSrcStep);
  Npp32f *dst_row = (Npp32f *)((char *)pDst + y * nDstStep);

  // Copy all three channels
  dst_row[x * 3 + 0] = src_row[x * 3 + 0]; // Channel 0 (R)
  dst_row[x * 3 + 1] = src_row[x * 3 + 1]; // Channel 1 (G)
  dst_row[x * 3 + 2] = src_row[x * 3 + 2]; // Channel 2 (B)
}

// Kernel for 32-bit float packed to planar copy (C3P3R = packed to planar)
__global__ void nppiCopy_32f_C3P3R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *const *pDst, int nDstStep,
                                          int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  // Read from packed source (RGB interleaved)
  const Npp32f *src_row = (const Npp32f *)((const char *)pSrc + y * nSrcStep);

  // Write to planar destinations (separate channels)
  for (int c = 0; c < 3; c++) {
    Npp32f *dst_row = (Npp32f *)((char *)pDst[c] + y * nDstStep);
    dst_row[x] = src_row[x * 3 + c]; // Extract channel c from packed format
  }
}

// Optimized kernel using vectorized loads for better memory throughput
__global__ void nppiCopy_8u_C4R_vectorized_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                                  int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const uint32_t *src_row = (const uint32_t *)((const char *)pSrc + y * nSrcStep);
  uint32_t *dst_row = (uint32_t *)((char *)pDst + y * nDstStep);

  // Copy 4 bytes (1 pixel) at once
  dst_row[x] = src_row[x];
}

extern "C" {

// 8-bit unsigned single channel copy implementation
NppStatus nppiCopy_8u_C1R_Ctx_cuda(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_8u_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                           oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// 8-bit unsigned three channel copy implementation
NppStatus nppiCopy_8u_C3R_Ctx_cuda(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_8u_C3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                           oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// 8-bit unsigned four channel copy implementation
NppStatus nppiCopy_8u_C4R_Ctx_cuda(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                   NppStreamContext nppStreamCtx) {
  // Use vectorized version if width is aligned to 4-byte boundaries
  if (oSizeROI.width % 4 == 0 && nSrcStep % 4 == 0 && nDstStep % 4 == 0) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

    nppiCopy_8u_C4R_vectorized_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pDst, nDstStep, oSizeROI.width, oSizeROI.height);
  } else {
    // Fall back to regular implementation
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

    nppiCopy_8u_C4R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                             oSizeROI.width, oSizeROI.height);
  }

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// 32-bit float single channel copy implementation
NppStatus nppiCopy_32f_C1R_Ctx_cuda(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_32f_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                            oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// 32-bit float three channel copy implementation
NppStatus nppiCopy_32f_C3R_Ctx_cuda(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                    NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_32f_C3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                            oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// 32-bit float packed to planar copy implementation
NppStatus nppiCopy_32f_C3P3R_Ctx_cuda(const Npp32f *pSrc, int nSrcStep, Npp32f *const pDst[3], int nDstStep,
                                      NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  // Copy destination pointers to device memory for kernel access
  Npp32f **d_pDst;

  hipError_t cudaStatus = hipMalloc(&d_pDst, 3 * sizeof(Npp32f *));
  if (cudaStatus != hipSuccess) {
    return NPP_MEMORY_ALLOCATION_ERR;
  }

  cudaStatus = hipMemcpy(d_pDst, pDst, 3 * sizeof(Npp32f *), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    hipFree(d_pDst);
    return NPP_MEMORY_ALLOCATION_ERR;
  }

  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiCopy_32f_C3P3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, d_pDst, nDstStep,
                                                                              oSizeROI.width, oSizeROI.height);

  cudaStatus = hipGetLastError();

  // Clean up device memory
  hipFree(d_pDst);

  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}
}