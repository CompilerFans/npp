#include "npp.h"
#include <hip/hip_runtime.h>


// Implementation file

// Implementation file
__global__ void convert_8u32f_C1R_kernel(const Npp8u *__restrict__ pSrc, int nSrcStep, Npp32f *__restrict__ pDst,
                                         int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *srcRow = (const Npp8u *)(((const char *)pSrc) + y * nSrcStep);
  Npp32f *dstRow = (Npp32f *)(((char *)pDst) + y * nDstStep);

  // Convert 8-bit unsigned to 32-bit float (0-255 -> 0.0-255.0)
  dstRow[x] = (Npp32f)srcRow[x];
}

// Implementation file
__global__ void convert_8u32f_C3R_kernel(const Npp8u *__restrict__ pSrc, int nSrcStep, Npp32f *__restrict__ pDst,
                                         int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *srcRow = (const Npp8u *)(((const char *)pSrc) + y * nSrcStep);
  Npp32f *dstRow = (Npp32f *)(((char *)pDst) + y * nDstStep);

  // Convert all three channels: 8-bit unsigned to 32-bit float (0-255 -> 0.0-255.0)
  dstRow[x * 3 + 0] = (Npp32f)srcRow[x * 3 + 0]; // Channel 0 (R/B)
  dstRow[x * 3 + 1] = (Npp32f)srcRow[x * 3 + 1]; // Channel 1 (G)
  dstRow[x * 3 + 2] = (Npp32f)srcRow[x * 3 + 2]; // Channel 2 (B/R)
}

extern "C" {

// Implementation file
NppStatus nppiConvert_8u32f_C1R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                         NppStreamContext nppStreamCtx) {

  // Setup kernel launch parameters
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  // Launch kernel with the specified GPU stream
  convert_8u32f_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                             oSizeROI.width, oSizeROI.height);

  // Check for kernel launch errors
  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_NO_ERROR;
}

// Implementation file
NppStatus nppiConvert_8u32f_C3R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, NppiSize oSizeROI,
                                         NppStreamContext nppStreamCtx) {

  // Parameter validation
  if (!pSrc || !pDst)
    return NPP_NULL_POINTER_ERROR;
  if (nDstStep <= 0)
    return NPP_STEP_ERROR;
  if (nSrcStep <= 0)
    return NPP_NO_ERROR;
  if (oSizeROI.width < 0 || oSizeROI.height < 0)
    return NPP_SIZE_ERROR;
  if (oSizeROI.width == 0 || oSizeROI.height == 0)
    return NPP_NO_ERROR;

  // Setup kernel launch parameters
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  // Launch kernel with the specified GPU stream
  convert_8u32f_C3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                             oSizeROI.width, oSizeROI.height);

  // Check for kernel launch errors
  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_NO_ERROR;
}

} // extern "C"