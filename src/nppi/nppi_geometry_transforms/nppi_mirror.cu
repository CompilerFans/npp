#include "nppdefs.h"
#include <hip/hip_runtime.h>


// Mirror flip kernel implementation
__global__ void nppiMirror_8u_C1R_kernel_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                              int height, NppiAxis flip) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int srcX = x;
    int srcY = y;
    int dstX = x;
    int dstY = y;

    // Compute source coordinates based on flip axis
    if (flip == NPP_HORIZONTAL_AXIS) {
      // Horizontal flip (top-bottom flip)
      srcY = height - 1 - y;
    } else if (flip == NPP_VERTICAL_AXIS) {
      // Vertical flip (left-right flip)
      srcX = width - 1 - x;
    } else if (flip == NPP_BOTH_AXIS) {
      // Dual axis flip (180 degree rotation)
      srcX = width - 1 - x;
      srcY = height - 1 - y;
    }

    // Read source pixel and write to destination
    const Npp8u *pSrcRow = (const Npp8u *)((const char *)pSrc + srcY * nSrcStep);
    Npp8u *pDstRow = (Npp8u *)((char *)pDst + dstY * nDstStep);

    pDstRow[dstX] = pSrcRow[srcX];
  }
}

extern "C" {
hipError_t nppiMirror_8u_C1R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oROI,
                                     NppiAxis flip, hipStream_t stream) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oROI.width + blockSize.x - 1) / blockSize.x, (oROI.height + blockSize.y - 1) / blockSize.y);

  nppiMirror_8u_C1R_kernel_impl<<<gridSize, blockSize, 0, stream>>>(pSrc, nSrcStep, pDst, nDstStep, oROI.width,
                                                                    oROI.height, flip);

  return hipGetLastError();
}
}
