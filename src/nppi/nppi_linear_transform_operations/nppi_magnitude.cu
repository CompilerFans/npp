#include "npp.h"
#include <cmath>
#include <hip/hip_runtime.h>


/**
 * CUDA kernels for NPP Image Magnitude operations
 * Implements magnitude and magnitude squared for complex numbers
 */

/**
 * CUDA kernel for computing magnitude from complex numbers
 * magnitude = sqrt(real^2 + imag^2)
 */
__global__ void magnitude_32fc32f_kernel(const Npp32fc *__restrict__ pSrc, int nSrcStep, Npp32f *__restrict__ pDst,
                                         int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp32fc *srcPtr = (const Npp32fc *)((const char *)pSrc + y * nSrcStep);
    Npp32f *dstPtr = (Npp32f *)((char *)pDst + y * nDstStep);

    Npp32fc srcValue = srcPtr[x];
    float real = srcValue.re;
    float imag = srcValue.im;

    // Compute magnitude: sqrt(real^2 + imag^2)
    dstPtr[x] = sqrtf(real * real + imag * imag);
  }
}

/**
 * CUDA kernel for computing squared magnitude from complex numbers
 * magnitude_sqr = real^2 + imag^2
 */
__global__ void magnitude_sqr_32fc32f_kernel(const Npp32fc *__restrict__ pSrc, int nSrcStep, Npp32f *__restrict__ pDst,
                                             int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp32fc *srcPtr = (const Npp32fc *)((const char *)pSrc + y * nSrcStep);
    Npp32f *dstPtr = (Npp32f *)((char *)pDst + y * nDstStep);

    Npp32fc srcValue = srcPtr[x];
    float real = srcValue.re;
    float imag = srcValue.im;

    // Compute squared magnitude: real^2 + imag^2
    dstPtr[x] = real * real + imag * imag;
  }
}

extern "C" {

// ============================================================================
// Magnitude function implementations
// ============================================================================

NppStatus nppiMagnitude_32fc32f_C1R_Ctx_cuda(const Npp32fc *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep,
                                             NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  magnitude_32fc32f_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                             oSizeROI.width, oSizeROI.height);

  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_NO_ERROR;
}

NppStatus nppiMagnitudeSqr_32fc32f_C1R_Ctx_cuda(const Npp32fc *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep,
                                                NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  magnitude_sqr_32fc32f_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                 oSizeROI.width, oSizeROI.height);

  hipError_t cudaErr = hipGetLastError();
  if (cudaErr != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_NO_ERROR;
}

} // extern "C"