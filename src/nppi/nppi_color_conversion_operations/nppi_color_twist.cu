#include "npp.h"
#include <hip/hip_runtime.h>


/**
 * CUDA kernels for NPP Color Twist Functions
 */

// Color twist kernel for 3-channel 8-bit images
__global__ void nppiColorTwist32f_8u_C3R_kernel(const Npp8u* pSrc, int nSrcStep,
                                               Npp8u* pDst, int nDstStep,
                                               int width, int height,
                                               const float* twist) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) {
        return;
    }
    
    const Npp8u* src_row = (const Npp8u*)((const char*)pSrc + y * nSrcStep);
    Npp8u* dst_row = (Npp8u*)((char*)pDst + y * nDstStep);
    
    int src_idx = x * 3;
    int dst_idx = x * 3;
    
    // Get source RGB values
    float r = (float)src_row[src_idx + 0];
    float g = (float)src_row[src_idx + 1];
    float b = (float)src_row[src_idx + 2];
    
    // Apply color twist matrix: [R' G' B']^T = [twist] * [R G B 1]^T
    float r_new = twist[0] * r + twist[1] * g + twist[2] * b + twist[3];
    float g_new = twist[4] * r + twist[5] * g + twist[6] * b + twist[7];
    float b_new = twist[8] * r + twist[9] * g + twist[10] * b + twist[11];
    
    // Clamp and store results
    dst_row[dst_idx + 0] = (Npp8u)fmaxf(0.0f, fminf(255.0f, r_new + 0.5f));
    dst_row[dst_idx + 1] = (Npp8u)fmaxf(0.0f, fminf(255.0f, g_new + 0.5f));
    dst_row[dst_idx + 2] = (Npp8u)fmaxf(0.0f, fminf(255.0f, b_new + 0.5f));
}

// Color twist kernel for single-channel 8-bit images (grayscale)
__global__ void nppiColorTwist32f_8u_C1R_kernel(const Npp8u* pSrc, int nSrcStep,
                                               Npp8u* pDst, int nDstStep,
                                               int width, int height,
                                               const float* twist) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) {
        return;
    }
    
    const Npp8u* src_row = (const Npp8u*)((const char*)pSrc + y * nSrcStep);
    Npp8u* dst_row = (Npp8u*)((char*)pDst + y * nDstStep);
    
    // Get source grayscale value
    float gray = (float)src_row[x];
    
    // Apply first row of color twist matrix (treat as RGB->R transformation)
    float result = twist[0] * gray + twist[1] * gray + twist[2] * gray + twist[3];
    
    // Clamp and store result
    dst_row[x] = (Npp8u)fmaxf(0.0f, fminf(255.0f, result + 0.5f));
}

extern "C" {

// 3-channel 8-bit implementation
NppStatus nppiColorTwist32f_8u_C3R_Ctx_cuda(const Npp8u* pSrc, int nSrcStep,
                                           Npp8u* pDst, int nDstStep,
                                           NppiSize oSizeROI,
                                           const Npp32f aTwist[3][4],
                                           NppStreamContext nppStreamCtx) {
    // Copy twist matrix to device memory
    float* d_twist;
    hipError_t cudaStatus = hipMalloc(&d_twist, 12 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        return NPP_MEMORY_ALLOCATION_ERR;
    }
    
    // Flatten 3x4 matrix to 1D array
    float h_twist[12];
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 4; j++) {
            h_twist[i * 4 + j] = aTwist[i][j];
        }
    }
    
    cudaStatus = hipMemcpyAsync(d_twist, h_twist, 12 * sizeof(float),
                                hipMemcpyHostToDevice, nppStreamCtx.hStream);
    if (cudaStatus != hipSuccess) {
        hipFree(d_twist);
        return NPP_MEMORY_ALLOCATION_ERR;
    }
    
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiColorTwist32f_8u_C3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pDst, nDstStep, oSizeROI.width, oSizeROI.height, d_twist);
    
    cudaStatus = hipGetLastError();
    
    // Sync stream and free memory
    hipStreamSynchronize(nppStreamCtx.hStream);
    hipFree(d_twist);
    
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

// Single-channel 8-bit implementation
NppStatus nppiColorTwist32f_8u_C1R_Ctx_cuda(const Npp8u* pSrc, int nSrcStep,
                                           Npp8u* pDst, int nDstStep,
                                           NppiSize oSizeROI,
                                           const Npp32f aTwist[3][4],
                                           NppStreamContext nppStreamCtx) {
    // Copy twist matrix to device memory
    float* d_twist;
    hipError_t cudaStatus = hipMalloc(&d_twist, 12 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        return NPP_MEMORY_ALLOCATION_ERR;
    }
    
    // Flatten 3x4 matrix to 1D array
    float h_twist[12];
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 4; j++) {
            h_twist[i * 4 + j] = aTwist[i][j];
        }
    }
    
    cudaStatus = hipMemcpyAsync(d_twist, h_twist, 12 * sizeof(float),
                                hipMemcpyHostToDevice, nppStreamCtx.hStream);
    if (cudaStatus != hipSuccess) {
        hipFree(d_twist);
        return NPP_MEMORY_ALLOCATION_ERR;
    }
    
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiColorTwist32f_8u_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pDst, nDstStep, oSizeROI.width, oSizeROI.height, d_twist);
    
    cudaStatus = hipGetLastError();
    
    // Sync stream and free memory
    hipStreamSynchronize(nppStreamCtx.hStream);
    hipFree(d_twist);
    
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

}