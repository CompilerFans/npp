#include "npp.h"
#include <hip/hip_runtime.h>


// Implementation file

// ITU-R BT.709 standard luminance weights
#define WEIGHT_R 0.299f
#define WEIGHT_G 0.587f
#define WEIGHT_B 0.114f

// Implementation file
__global__ void nppiRGBToGray_8u_C3C1R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                              int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    // Calculate byte offsets correctly
    const Npp8u *src_pixel = pSrc + y * nSrcStep + x * 3;
    Npp8u *dst_pixel = pDst + y * nDstStep + x;

    // Get RGB values
    Npp8u r = src_pixel[0];
    Npp8u g = src_pixel[1];
    Npp8u b = src_pixel[2];

    // Calculate grayscale using ITU-R BT.709 weights
    float gray = WEIGHT_R * r + WEIGHT_G * g + WEIGHT_B * b;
    *dst_pixel = (Npp8u)(gray + 0.5f); // Round to nearest
  }
}

// Implementation file
__global__ void nppiRGBToGray_8u_AC4C1R_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, int width,
                                               int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    // Calculate byte offsets correctly
    const Npp8u *src_pixel = pSrc + y * nSrcStep + x * 4;
    Npp8u *dst_pixel = pDst + y * nDstStep + x;

    // Get RGB values (ignore alpha channel)
    Npp8u r = src_pixel[0];
    Npp8u g = src_pixel[1];
    Npp8u b = src_pixel[2];
    // Alpha channel at index 3 is ignored

    // Calculate grayscale using ITU-R BT.709 weights
    float gray = WEIGHT_R * r + WEIGHT_G * g + WEIGHT_B * b;
    *dst_pixel = (Npp8u)(gray + 0.5f); // Round to nearest
  }
}

// Implementation file
__global__ void nppiRGBToGray_32f_C3C1R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, int width,
                                               int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    // Calculate byte offsets correctly for 32-bit float
    const Npp32f *src_pixel = (const Npp32f *)((const char *)pSrc + y * nSrcStep) + x * 3;
    Npp32f *dst_pixel = (Npp32f *)((char *)pDst + y * nDstStep) + x;

    // Get RGB values
    Npp32f r = src_pixel[0];
    Npp32f g = src_pixel[1];
    Npp32f b = src_pixel[2];

    // Calculate grayscale using ITU-R BT.709 weights
    *dst_pixel = WEIGHT_R * r + WEIGHT_G * g + WEIGHT_B * b;
  }
}

// Implementation file
__global__ void nppiRGBToGray_32f_AC4C1R_kernel(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep, int width,
                                                int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    // Calculate byte offsets correctly for 32-bit float
    const Npp32f *src_pixel = (const Npp32f *)((const char *)pSrc + y * nSrcStep) + x * 4;
    Npp32f *dst_pixel = (Npp32f *)((char *)pDst + y * nDstStep) + x;

    // Get RGB values (ignore alpha channel)
    Npp32f r = src_pixel[0];
    Npp32f g = src_pixel[1];
    Npp32f b = src_pixel[2];
    // Alpha channel at index 3 is ignored

    // Calculate grayscale using ITU-R BT.709 weights
    *dst_pixel = WEIGHT_R * r + WEIGHT_G * g + WEIGHT_B * b;
  }
}

extern "C" {

// Implementation file
NppStatus nppiRGBToGray_8u_C3C1R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI,
                                          NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiRGBToGray_8u_C3C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                  oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  // Synchronize to ensure kernel completion
  if (nppStreamCtx.hStream == 0) {
    cudaStatus = hipDeviceSynchronize();
  } else {
    cudaStatus = hipStreamSynchronize(nppStreamCtx.hStream);
  }
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// Implementation file
NppStatus nppiRGBToGray_8u_AC4C1R_Ctx_impl(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep,
                                           NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiRGBToGray_8u_AC4C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                   oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  // Synchronize to ensure kernel completion
  if (nppStreamCtx.hStream == 0) {
    cudaStatus = hipDeviceSynchronize();
  } else {
    cudaStatus = hipStreamSynchronize(nppStreamCtx.hStream);
  }
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// Implementation file
NppStatus nppiRGBToGray_32f_C3C1R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep,
                                           NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiRGBToGray_32f_C3C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                   oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  // Synchronize to ensure kernel completion
  if (nppStreamCtx.hStream == 0) {
    cudaStatus = hipDeviceSynchronize();
  } else {
    cudaStatus = hipStreamSynchronize(nppStreamCtx.hStream);
  }
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

// Implementation file
NppStatus nppiRGBToGray_32f_AC4C1R_Ctx_impl(const Npp32f *pSrc, int nSrcStep, Npp32f *pDst, int nDstStep,
                                            NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 blockSize(16, 16);
  dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x, (oSizeROI.height + blockSize.y - 1) / blockSize.y);

  nppiRGBToGray_32f_AC4C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrc, nSrcStep, pDst, nDstStep,
                                                                                    oSizeROI.width, oSizeROI.height);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  // Synchronize to ensure kernel completion
  if (nppStreamCtx.hStream == 0) {
    cudaStatus = hipDeviceSynchronize();
  } else {
    cudaStatus = hipStreamSynchronize(nppStreamCtx.hStream);
  }
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}

} // extern "C"
