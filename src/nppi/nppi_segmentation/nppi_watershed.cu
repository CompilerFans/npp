#include "hip/hip_runtime.h"
#include "npp.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define WATERSHED_MASK -2
#define WATERSHED_WSHED -1
#define WATERSHED_INIT 0

// Pixel structure for priority queue
struct WatershedPixel {
  int x, y;
  Npp8u intensity;

  __device__ bool operator<(const WatershedPixel &other) const { return intensity < other.intensity; }
};

// Compute gradient magnitude
__global__ void computeGradient_kernel(const Npp8u *pSrc, int nSrcStep, Npp8u *pGradient, int nGradStep, int width,
                                       int height, Npp8u eNorm) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp8u *src_row = (const Npp8u *)((const char *)pSrc + y * nSrcStep);
  Npp8u *grad_row = (Npp8u *)((char *)pGradient + y * nGradStep);

  float gx = 0.0f, gy = 0.0f;

  // Compute gradient using Sobel operator
  if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
    const Npp8u *prev_row = (const Npp8u *)((const char *)pSrc + (y - 1) * nSrcStep);
    const Npp8u *next_row = (const Npp8u *)((const char *)pSrc + (y + 1) * nSrcStep);

    // X direction gradient
    gx = (float)prev_row[x + 1] + 2.0f * src_row[x + 1] + (float)next_row[x + 1] - (float)prev_row[x - 1] -
         2.0f * src_row[x - 1] - (float)next_row[x - 1];

    // Y direction gradient
    gy = (float)next_row[x - 1] + 2.0f * next_row[x] + (float)next_row[x + 1] - (float)prev_row[x - 1] -
         2.0f * prev_row[x] - (float)prev_row[x + 1];
  }

  float magnitude;
  if (eNorm == 1) {
    magnitude = fabs(gx) + fabs(gy); // L1 norm
  } else {
    magnitude = sqrtf(gx * gx + gy * gy); // L2 norm
  }

  grad_row[x] = (Npp8u)min(magnitude, 255.0f);
}

// Initialize markers

__global__ void initializeMarkers_kernel(Npp32s *pMarkers, int nMarkersStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  Npp32s *marker_row = (Npp32s *)((char *)pMarkers + y * nMarkersStep);

  if (marker_row[x] > 0) {
    // Keep marked seed points unchanged
    return;
  } else {
    // Initialize unmarked points as MASK
    marker_row[x] = WATERSHED_MASK;
  }
}

// Find boundary pixels (neighbors of seed points)
__global__ void findBoundaryPixels_kernel(const Npp32s *pMarkers, int nMarkersStep, const Npp8u *pGradient,
                                          int nGradStep, WatershedPixel *pBoundaryPixels, int *pBoundaryCount,
                                          int width, int height, int maxPixels) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  const Npp32s *marker_row = (const Npp32s *)((const char *)pMarkers + y * nMarkersStep);
  const Npp8u *grad_row = (const Npp8u *)((const char *)pGradient + y * nGradStep);

  // If current pixel is MASK, check if adjacent to marked region
  if (marker_row[x] == WATERSHED_MASK) {
    bool isBoundary = false;

    // Check 4-connected neighborhood
    int dx[] = {-1, 1, 0, 0};
    int dy[] = {0, 0, -1, 1};

    for (int i = 0; i < 4; i++) {
      int nx = x + dx[i];
      int ny = y + dy[i];

      if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
        const Npp32s *neighbor_row = (const Npp32s *)((const char *)pMarkers + ny * nMarkersStep);
        if (neighbor_row[nx] > 0) { // Neighbor is marked seed point
          isBoundary = true;
          break;
        }
      }
    }

    if (isBoundary) {
      int idx = atomicAdd(pBoundaryCount, 1);
      if (idx < maxPixels) {
        pBoundaryPixels[idx].x = x;
        pBoundaryPixels[idx].y = y;
        pBoundaryPixels[idx].intensity = grad_row[x];
      }
    }
  }
}

// Process pixels in queue
__global__ void processWatershedPixels_kernel(const WatershedPixel *pPixels, int pixelCount, Npp32s *pMarkers,
                                              int nMarkersStep, int width, int height, WatershedPixel *pNewPixels,
                                              int *pNewCount, const Npp8u *pGradient, int nGradStep) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= pixelCount)
    return;

  WatershedPixel pixel = pPixels[idx];
  int x = pixel.x;
  int y = pixel.y;

  Npp32s *marker_row = (Npp32s *)((char *)pMarkers + y * nMarkersStep);

  if (marker_row[x] != WATERSHED_MASK)
    return; // Already processed

  // Check neighborhood labels
  int dx[] = {-1, 1, 0, 0};
  int dy[] = {0, 0, -1, 1};

  Npp32s neighborLabel = 0;
  bool hasMultipleLabels = false;

  for (int i = 0; i < 4; i++) {
    int nx = x + dx[i];
    int ny = y + dy[i];

    if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
      const Npp32s *neighbor_row = (const Npp32s *)((const char *)pMarkers + ny * nMarkersStep);
      Npp32s label = neighbor_row[nx];

      if (label > 0) { // Valid label
        if (neighborLabel == 0) {
          neighborLabel = label;
        } else if (neighborLabel != label) {
          hasMultipleLabels = true;
          break;
        }
      }
    }
  }

  if (hasMultipleLabels) {
    marker_row[x] = WATERSHED_WSHED; // Watershed line
  } else if (neighborLabel > 0) {
    marker_row[x] = neighborLabel; // Extend label

    // Add MASK pixels in neighborhood to next processing round
    for (int i = 0; i < 4; i++) {
      int nx = x + dx[i];
      int ny = y + dy[i];

      if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
        const Npp32s *neighbor_row = (const Npp32s *)((const char *)pMarkers + ny * nMarkersStep);
        if (neighbor_row[nx] == WATERSHED_MASK) {
          int newIdx = atomicAdd(pNewCount, 1);
          if (newIdx < width * height) { // Prevent overflow
            const Npp8u *neighbor_grad_row = (const Npp8u *)((const char *)pGradient + ny * nGradStep);
            pNewPixels[newIdx].x = nx;
            pNewPixels[newIdx].y = ny;
            pNewPixels[newIdx].intensity = neighbor_grad_row[nx];
          }
        }
      }
    }
  }
}

extern "C" {

// Get required buffer size

NppStatus nppiSegmentWatershedGetBufferSize_8u_C1R_Ctx_impl(NppiSize oSizeROI, size_t *hpBufferSize) {
  size_t imageSize = (size_t)oSizeROI.width * oSizeROI.height;

  // Required buffers:
  // 1. Gradient image (Npp8u)
  // 2. Pixel queue 1 (WatershedPixel)
  // 3. Pixel queue 2 (WatershedPixel)
  // 4. Counter (int)

  size_t gradientSize = imageSize * sizeof(Npp8u);
  size_t queueSize = imageSize * sizeof(WatershedPixel) * 2;
  size_t counterSize = sizeof(int) * 2;

  size_t totalSize = gradientSize + queueSize + counterSize;
  size_t alignedSize = (totalSize + 511) & ~511; // 512byte alignment

  *hpBufferSize = alignedSize;
  return NPP_SUCCESS;
}

// WatershedSegmentation main function
NppStatus nppiSegmentWatershed_8u_C1IR_Ctx_impl(Npp8u *pSrcDst, Npp32s nSrcDstStep, Npp32u *pMarkerLabels,
                                                Npp32s nMarkerLabelsStep, NppiNorm eNorm, NppiSize oSizeROI,
                                                Npp8u *pDeviceBuffer, NppStreamContext nppStreamCtx) {
  int width = oSizeROI.width;
  int height = oSizeROI.height;
  size_t imageSize = width * height;

  // Setup buffers
  Npp8u *pGradient = pDeviceBuffer;
  WatershedPixel *pPixelQueue1 = (WatershedPixel *)(pGradient + imageSize);
  WatershedPixel *pPixelQueue2 = pPixelQueue1 + imageSize;
  int *pQueueCount1 = (int *)(pPixelQueue2 + imageSize);
  int *pQueueCount2 = pQueueCount1 + 1;

  int gradStep = width * sizeof(Npp8u);

  dim3 blockSize(16, 16);
  dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

  // Step 1: Compute gradient
  computeGradient_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(pSrcDst, nSrcDstStep, pGradient, gradStep,
                                                                           width, height, (Npp8u)eNorm);

  // Step 2: Initialize markers
  initializeMarkers_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>((Npp32s *)pMarkerLabels, nMarkerLabelsStep,
                                                                             width, height);

  // Step 3: Find initial boundary pixels
  hipMemsetAsync(pQueueCount1, 0, sizeof(int), nppStreamCtx.hStream);

  findBoundaryPixels_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
      (Npp32s *)pMarkerLabels, nMarkerLabelsStep, pGradient, gradStep, pPixelQueue1, pQueueCount1, width, height,
      (int)imageSize);

  // Step 4: Iterative processing (simplified priority queue)
  WatershedPixel *currentQueue = pPixelQueue1;
  WatershedPixel *nextQueue = pPixelQueue2;
  int *currentCount = pQueueCount1;
  int *nextCount = pQueueCount2;

  for (int iteration = 0; iteration < 255; iteration++) { // Maximum 255 iterations
    int h_count;
    hipMemcpyAsync(&h_count, currentCount, sizeof(int), hipMemcpyDeviceToHost, nppStreamCtx.hStream);
    hipStreamSynchronize(nppStreamCtx.hStream);

    if (h_count == 0)
      break; // No more pixels to process

    // Clear next queue
    hipMemsetAsync(nextCount, 0, sizeof(int), nppStreamCtx.hStream);

    // Process all pixels at current intensity level
    dim3 linearBlockSize(256);
    dim3 linearGridSize((h_count + linearBlockSize.x - 1) / linearBlockSize.x);

    processWatershedPixels_kernel<<<linearGridSize, linearBlockSize, 0, nppStreamCtx.hStream>>>(
        currentQueue, h_count, (Npp32s *)pMarkerLabels, nMarkerLabelsStep, width, height, nextQueue, nextCount,
        pGradient, gradStep);

    // Swap queues
    WatershedPixel *tempQueue = currentQueue;
    currentQueue = nextQueue;
    nextQueue = tempQueue;

    int *tempCount = currentCount;
    currentCount = nextCount;
    nextCount = tempCount;
  }

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    return NPP_CUDA_KERNEL_EXECUTION_ERROR;
  }

  return NPP_SUCCESS;
}
}
