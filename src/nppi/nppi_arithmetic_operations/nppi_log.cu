#include "npp.h"
#include <hip/hip_runtime.h>

#include <cmath>

/**
 * CUDA kernels for NPP Common Logarithm Operations
 * Computes base-10 logarithm of input image values
 */

/**
 * CUDA kernel for 32-bit float common logarithm (base 10)
 */
__global__ void nppiLog_32f_C1R_kernel(const Npp32f* pSrc, int nSrcStep,
                                        Npp32f* pDst, int nDstStep,
                                        int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        const Npp32f* src_pixel = (const Npp32f*)((const char*)pSrc + y * nSrcStep) + x;
        Npp32f* dst_pixel = (Npp32f*)((char*)pDst + y * nDstStep) + x;
        
        float src_val = *src_pixel;
        
        // 计算以10为底的对数
        float result;
        if (src_val <= 0.0f) {
            // 负数或0的对数未定义，设为负无穷
            result = -HUGE_VALF;
        } else if (src_val == 1.0f) {
            // log10(1) = 0，特殊优化
            result = 0.0f;
        } else if (src_val == 10.0f) {
            // log10(10) = 1，特殊优化
            result = 1.0f;
        } else {
            // 一般情况：log10(x) = ln(x) / ln(10)
            result = log10f(src_val);
        }
        
        *dst_pixel = result;
    }
}

extern "C" {

/**
 * 32-bit float common logarithm (base 10)
 */
NppStatus nppiLog_32f_C1R_Ctx_cuda(const Npp32f* pSrc, int nSrcStep, Npp32f* pDst, int nDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiLog_32f_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pDst, nDstStep, oSizeROI.width, oSizeROI.height);
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    // 同步等待内核完成
    if (nppStreamCtx.hStream == 0) {
        cudaStatus = hipDeviceSynchronize();
    } else {
        cudaStatus = hipStreamSynchronize(nppStreamCtx.hStream);
    }
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

} // extern "C"