#include "hip/hip_runtime.h"
#include "nppi_arithmetic_executor.h"
#include "nppi_arithmetic_ops.h"

using namespace nppi::arithmetic;

// ============================================================================
// MaxEvery Implementation  
// nppiMaxEvery: result = max(src1, src2) for each pixel
// ============================================================================

extern "C" {

// 8u versions
NppStatus nppiMaxEvery_8u_C1IR_Ctx(const Npp8u *pSrc, int nSrcStep, Npp8u *pSrcDst, int nSrcDstStep,
                                   NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp8u, 1, MaxEveryOp<Npp8u>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMaxEvery_8u_C1IR(const Npp8u *pSrc, int nSrcStep, Npp8u *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMaxEvery_8u_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 16u versions
NppStatus nppiMaxEvery_16u_C1IR_Ctx(const Npp16u *pSrc, int nSrcStep, Npp16u *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp16u, 1, MaxEveryOp<Npp16u>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMaxEvery_16u_C1IR(const Npp16u *pSrc, int nSrcStep, Npp16u *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMaxEvery_16u_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 16s versions
NppStatus nppiMaxEvery_16s_C1IR_Ctx(const Npp16s *pSrc, int nSrcStep, Npp16s *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp16s, 1, MaxEveryOp<Npp16s>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMaxEvery_16s_C1IR(const Npp16s *pSrc, int nSrcStep, Npp16s *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMaxEvery_16s_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 32s versions
NppStatus nppiMaxEvery_32s_C1IR_Ctx(const Npp32s *pSrc, int nSrcStep, Npp32s *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp32s, 1, MaxEveryOp<Npp32s>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMaxEvery_32s_C1IR(const Npp32s *pSrc, int nSrcStep, Npp32s *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMaxEvery_32s_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 32f versions
NppStatus nppiMaxEvery_32f_C1IR_Ctx(const Npp32f *pSrc, int nSrcStep, Npp32f *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp32f, 1, MaxEveryOp<Npp32f>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMaxEvery_32f_C1IR(const Npp32f *pSrc, int nSrcStep, Npp32f *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMaxEvery_32f_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

} // extern "C"