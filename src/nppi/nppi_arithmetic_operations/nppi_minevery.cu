#include "hip/hip_runtime.h"
#include "nppi_arithmetic_executor.h"
#include "nppi_arithmetic_ops.h"

using namespace nppi::arithmetic;

// ============================================================================
// MinEvery Implementation  
// nppiMinEvery: result = min(src1, src2) for each pixel
// ============================================================================

extern "C" {

// 8u versions
NppStatus nppiMinEvery_8u_C1IR_Ctx(const Npp8u *pSrc, int nSrcStep, Npp8u *pSrcDst, int nSrcDstStep,
                                   NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp8u, 1, MinEveryOp<Npp8u>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMinEvery_8u_C1IR(const Npp8u *pSrc, int nSrcStep, Npp8u *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMinEvery_8u_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 16u versions
NppStatus nppiMinEvery_16u_C1IR_Ctx(const Npp16u *pSrc, int nSrcStep, Npp16u *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp16u, 1, MinEveryOp<Npp16u>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMinEvery_16u_C1IR(const Npp16u *pSrc, int nSrcStep, Npp16u *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMinEvery_16u_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 16s versions
NppStatus nppiMinEvery_16s_C1IR_Ctx(const Npp16s *pSrc, int nSrcStep, Npp16s *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp16s, 1, MinEveryOp<Npp16s>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMinEvery_16s_C1IR(const Npp16s *pSrc, int nSrcStep, Npp16s *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMinEvery_16s_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 32s versions
NppStatus nppiMinEvery_32s_C1IR_Ctx(const Npp32s *pSrc, int nSrcStep, Npp32s *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp32s, 1, MinEveryOp<Npp32s>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMinEvery_32s_C1IR(const Npp32s *pSrc, int nSrcStep, Npp32s *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMinEvery_32s_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

// 32f versions
NppStatus nppiMinEvery_32f_C1IR_Ctx(const Npp32f *pSrc, int nSrcStep, Npp32f *pSrcDst, int nSrcDstStep,
                                    NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  return BinaryOperationExecutor<Npp32f, 1, MinEveryOp<Npp32f>>::execute(
      pSrc, nSrcStep, pSrcDst, nSrcDstStep, pSrcDst, nSrcDstStep, oSizeROI, 0, nppStreamCtx.hStream);
}

NppStatus nppiMinEvery_32f_C1IR(const Npp32f *pSrc, int nSrcStep, Npp32f *pSrcDst, int nSrcDstStep, NppiSize oSizeROI) {
  NppStreamContext defaultCtx = {};
  defaultCtx.hStream = 0; // Default CUDA stream
  return nppiMinEvery_32f_C1IR_Ctx(pSrc, nSrcStep, pSrcDst, nSrcDstStep, oSizeROI, defaultCtx);
}

} // extern "C"