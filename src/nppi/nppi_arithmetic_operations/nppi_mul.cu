#include "npp.h"
#include <cmath>
#include <hip/hip_runtime.h>




// ============================================================================
// Device kernels
// ============================================================================

// 8-bit unsigned mul with scaling, single channel
__global__ void nppiMul_8u_C1RSfs_kernel(const Npp8u *pSrc1, int nSrc1Step, const Npp8u *pSrc2, int nSrc2Step,
                                         Npp8u *pDst, int nDstStep, int width, int height, int nScaleFactor) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp8u *src1Row = (const Npp8u *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp8u *src2Row = (const Npp8u *)((const char *)pSrc2 + y * nSrc2Step);
    Npp8u *dstRow = (Npp8u *)((char *)pDst + y * nDstStep);

    // Multiply with scaling (pSrc1 * pSrc2)
    int result = (int)src1Row[x] * (int)src2Row[x];

    // Apply scale factor (right shift)
    if (nScaleFactor > 0) {
      result = (result + (1 << (nScaleFactor - 1))) >> nScaleFactor;
    }

    // Saturate to 8-bit range (clamp to 0-255)
    dstRow[x] = (Npp8u)(result < 0 ? 0 : (result > 255 ? 255 : result));
  }
}

// 8-bit unsigned mul with scaling, 3 channels
__global__ void nppiMul_8u_C3RSfs_kernel(const Npp8u *pSrc1, int nSrc1Step, const Npp8u *pSrc2, int nSrc2Step,
                                         Npp8u *pDst, int nDstStep, int width, int height, int nScaleFactor) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp8u *src1Row = (const Npp8u *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp8u *src2Row = (const Npp8u *)((const char *)pSrc2 + y * nSrc2Step);
    Npp8u *dstRow = (Npp8u *)((char *)pDst + y * nDstStep);

    int idx = x * 3;

    // Process 3 channels
    for (int c = 0; c < 3; c++) {
      int result = (int)src1Row[idx + c] * (int)src2Row[idx + c];

      // Apply scale factor
      if (nScaleFactor > 0) {
        result = (result + (1 << (nScaleFactor - 1))) >> nScaleFactor;
      }

      // Saturate to 8-bit range
      dstRow[idx + c] = (Npp8u)(result < 0 ? 0 : (result > 255 ? 255 : result));
    }
  }
}

// 16-bit unsigned mul with scaling, single channel
__global__ void nppiMul_16u_C1RSfs_kernel(const Npp16u *pSrc1, int nSrc1Step, const Npp16u *pSrc2, int nSrc2Step,
                                          Npp16u *pDst, int nDstStep, int width, int height, int nScaleFactor) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp16u *src1Row = (const Npp16u *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp16u *src2Row = (const Npp16u *)((const char *)pSrc2 + y * nSrc2Step);
    Npp16u *dstRow = (Npp16u *)((char *)pDst + y * nDstStep);

    // Multiply with scaling - use larger intermediate type
    long long result = (long long)src1Row[x] * (long long)src2Row[x];

    // Apply scale factor
    if (nScaleFactor > 0) {
      result = (result + (1LL << (nScaleFactor - 1))) >> nScaleFactor;
    }

    // Saturate to 16-bit range
    dstRow[x] = (Npp16u)(result < 0 ? 0 : (result > 65535 ? 65535 : result));
  }
}

// 16-bit signed mul with scaling, single channel
__global__ void nppiMul_16s_C1RSfs_kernel(const Npp16s *pSrc1, int nSrc1Step, const Npp16s *pSrc2, int nSrc2Step,
                                          Npp16s *pDst, int nDstStep, int width, int height, int nScaleFactor) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp16s *src1Row = (const Npp16s *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp16s *src2Row = (const Npp16s *)((const char *)pSrc2 + y * nSrc2Step);
    Npp16s *dstRow = (Npp16s *)((char *)pDst + y * nDstStep);

    // Multiply with scaling - use larger intermediate type
    long long result = (long long)src1Row[x] * (long long)src2Row[x];

    // Apply scale factor
    if (nScaleFactor > 0) {
      result = (result + (1LL << (nScaleFactor - 1))) >> nScaleFactor;
    }

    // Saturate to 16-bit signed range
    dstRow[x] = (Npp16s)(result < -32768 ? -32768 : (result > 32767 ? 32767 : result));
  }
}

// 32-bit float mul, single channel (no scaling)
__global__ void nppiMul_32f_C1R_kernel(const Npp32f *pSrc1, int nSrc1Step, const Npp32f *pSrc2, int nSrc2Step,
                                       Npp32f *pDst, int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp32f *src1Row = (const Npp32f *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp32f *src2Row = (const Npp32f *)((const char *)pSrc2 + y * nSrc2Step);
    Npp32f *dstRow = (Npp32f *)((char *)pDst + y * nDstStep);

    // Simple float multiplication
    dstRow[x] = src1Row[x] * src2Row[x];
  }
}

// 32-bit float mul, 3 channels (no scaling)
__global__ void nppiMul_32f_C3R_kernel(const Npp32f *pSrc1, int nSrc1Step, const Npp32f *pSrc2, int nSrc2Step,
                                       Npp32f *pDst, int nDstStep, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    const Npp32f *src1Row = (const Npp32f *)((const char *)pSrc1 + y * nSrc1Step);
    const Npp32f *src2Row = (const Npp32f *)((const char *)pSrc2 + y * nSrc2Step);
    Npp32f *dstRow = (Npp32f *)((char *)pDst + y * nDstStep);

    int idx = x * 3;

    // Process 3 channels
    dstRow[idx] = src1Row[idx] * src2Row[idx];             // R
    dstRow[idx + 1] = src1Row[idx + 1] * src2Row[idx + 1]; // G
    dstRow[idx + 2] = src1Row[idx + 2] * src2Row[idx + 2]; // B
  }
}

// ============================================================================
// Host functions
// ============================================================================

extern "C" {

// 8-bit unsigned
NppStatus nppiMul_8u_C1RSfs_Ctx_impl(const Npp8u *pSrc1, int nSrc1Step, const Npp8u *pSrc2, int nSrc2Step, Npp8u *pDst,
                                     int nDstStep, NppiSize oSizeROI, int nScaleFactor, NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_8u_C1RSfs_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep,
                                                                     oSizeROI.width, oSizeROI.height, nScaleFactor);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}

NppStatus nppiMul_8u_C3RSfs_Ctx_impl(const Npp8u *pSrc1, int nSrc1Step, const Npp8u *pSrc2, int nSrc2Step, Npp8u *pDst,
                                     int nDstStep, NppiSize oSizeROI, int nScaleFactor, NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_8u_C3RSfs_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep,
                                                                     oSizeROI.width, oSizeROI.height, nScaleFactor);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}

// 16-bit unsigned
NppStatus nppiMul_16u_C1RSfs_Ctx_impl(const Npp16u *pSrc1, int nSrc1Step, const Npp16u *pSrc2, int nSrc2Step,
                                      Npp16u *pDst, int nDstStep, NppiSize oSizeROI, int nScaleFactor,
                                      NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_16u_C1RSfs_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(
      pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep, oSizeROI.width, oSizeROI.height, nScaleFactor);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}

// 16-bit signed
NppStatus nppiMul_16s_C1RSfs_Ctx_impl(const Npp16s *pSrc1, int nSrc1Step, const Npp16s *pSrc2, int nSrc2Step,
                                      Npp16s *pDst, int nDstStep, NppiSize oSizeROI, int nScaleFactor,
                                      NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_16s_C1RSfs_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(
      pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep, oSizeROI.width, oSizeROI.height, nScaleFactor);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}

// 32-bit float
NppStatus nppiMul_32f_C1R_Ctx_impl(const Npp32f *pSrc1, int nSrc1Step, const Npp32f *pSrc2, int nSrc2Step, Npp32f *pDst,
                                   int nDstStep, NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_32f_C1R_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep,
                                                                   oSizeROI.width, oSizeROI.height);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}

NppStatus nppiMul_32f_C3R_Ctx_impl(const Npp32f *pSrc1, int nSrc1Step, const Npp32f *pSrc2, int nSrc2Step, Npp32f *pDst,
                                   int nDstStep, NppiSize oSizeROI, NppStreamContext nppStreamCtx) {
  dim3 block(16, 16);
  dim3 grid((oSizeROI.width + block.x - 1) / block.x, (oSizeROI.height + block.y - 1) / block.y);

  nppiMul_32f_C3R_kernel<<<grid, block, 0, nppStreamCtx.hStream>>>(pSrc1, nSrc1Step, pSrc2, nSrc2Step, pDst, nDstStep,
                                                                   oSizeROI.width, oSizeROI.height);

  return hipGetLastError() == hipSuccess ? NPP_NO_ERROR : NPP_CUDA_KERNEL_EXECUTION_ERROR;
}
}
