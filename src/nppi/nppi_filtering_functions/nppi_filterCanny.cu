#include "npp.h"
#include <hip/hip_runtime.h>

#include <cmath>

/**
 * CUDA kernels for Canny Edge Detection
 */

// Sobel X和Y方向的核
__constant__ float c_sobelX[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
__constant__ float c_sobelY[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

// 高斯滤波核(3x3)
__constant__ float c_gaussianKernel3x3[9] = {
    1.0f/16, 2.0f/16, 1.0f/16,
    2.0f/16, 4.0f/16, 2.0f/16,
    1.0f/16, 2.0f/16, 1.0f/16
};

// 高斯滤波核(5x5)
__constant__ float c_gaussianKernel5x5[25] = {
    1.0f/256,  4.0f/256,  6.0f/256,  4.0f/256, 1.0f/256,
    4.0f/256, 16.0f/256, 24.0f/256, 16.0f/256, 4.0f/256,
    6.0f/256, 24.0f/256, 36.0f/256, 24.0f/256, 6.0f/256,
    4.0f/256, 16.0f/256, 24.0f/256, 16.0f/256, 4.0f/256,
    1.0f/256,  4.0f/256,  6.0f/256,  4.0f/256, 1.0f/256
};

template<typename T>
__device__ T getBorderPixelCanny(const T* pSrc, int nSrcStep, int width, int height,
                                 int x, int y, NppiBorderType eBorderType) {
    if (x >= 0 && x < width && y >= 0 && y < height) {
        const T* src_row = (const T*)((const char*)pSrc + y * nSrcStep);
        return src_row[x];
    }
    
    switch (eBorderType) {
        case NPP_BORDER_REPLICATE:
            x = max(0, min(x, width - 1));
            y = max(0, min(y, height - 1));
            break;
        case NPP_BORDER_CONSTANT:
        default:
            return T(0);
    }
    
    const T* src_row = (const T*)((const char*)pSrc + y * nSrcStep);
    return src_row[x];
}

// 第一步：高斯滤波去噪
__global__ void cannyGaussianBlur_kernel(const Npp8u* pSrc, int nSrcStep, 
                                         float* pBlurred, int nBlurredStep,
                                         int srcWidth, int srcHeight,
                                         NppiMaskSize eMaskSize, NppiBorderType eBorderType) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= srcWidth || y >= srcHeight) return;
    
    float sum = 0.0f;
    int maskSize = (eMaskSize == NPP_MASK_SIZE_3_X_3) ? 3 : 5;
    int half = maskSize / 2;
    
    for (int dy = -half; dy <= half; dy++) {
        for (int dx = -half; dx <= half; dx++) {
            float pixel = (float)getBorderPixelCanny<Npp8u>(pSrc, nSrcStep, srcWidth, srcHeight,
                                                            x + dx, y + dy, eBorderType);
            
            if (eMaskSize == NPP_MASK_SIZE_3_X_3) {
                sum += pixel * c_gaussianKernel3x3[(dy + half) * 3 + (dx + half)];
            } else {
                sum += pixel * c_gaussianKernel5x5[(dy + half) * 5 + (dx + half)];
            }
        }
    }
    
    float* blurred_row = (float*)((char*)pBlurred + y * nBlurredStep);
    blurred_row[x] = sum;
}

// 第二步：计算梯度强度和方向
__global__ void cannySobelGradient_kernel(const float* pBlurred, int nBlurredStep,
                                          float* pGradMag, int nMagStep,
                                          float* pGradDir, int nDirStep,
                                          int width, int height, NppiBorderType eBorderType) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    float gx = 0.0f, gy = 0.0f;
    
    // 应用Sobel算子
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int px = x + dx;
            int py = y + dy;
            
            float pixel;
            if (px >= 0 && px < width && py >= 0 && py < height) {
                const float* blurred_row = (const float*)((const char*)pBlurred + py * nBlurredStep);
                pixel = blurred_row[px];
            } else {
                // 边界处理
                if (eBorderType == NPP_BORDER_REPLICATE) {
                    px = max(0, min(px, width - 1));
                    py = max(0, min(py, height - 1));
                    const float* blurred_row = (const float*)((const char*)pBlurred + py * nBlurredStep);
                    pixel = blurred_row[px];
                } else {
                    pixel = 0.0f;
                }
            }
            
            int kernelIdx = (dy + 1) * 3 + (dx + 1);
            gx += pixel * c_sobelX[kernelIdx];
            gy += pixel * c_sobelY[kernelIdx];
        }
    }
    
    // 计算梯度强度和方向
    float magnitude = sqrtf(gx * gx + gy * gy);
    float direction = atan2f(gy, gx) * 180.0f / M_PI;  // 转换为度数
    
    // 将方向标准化到0-180度
    if (direction < 0) direction += 180.0f;
    
    float* mag_row = (float*)((char*)pGradMag + y * nMagStep);
    float* dir_row = (float*)((char*)pGradDir + y * nDirStep);
    
    mag_row[x] = magnitude;
    dir_row[x] = direction;
}

// 第三步：非最大值抑制
__global__ void cannyNonMaxSuppression_kernel(const float* pGradMag, int nMagStep,
                                              const float* pGradDir, int nDirStep,
                                              float* pSuppressed, int nSuppressedStep,
                                              int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    const float* mag_row = (const float*)((const char*)pGradMag + y * nMagStep);
    const float* dir_row = (const float*)((const char*)pGradDir + y * nDirStep);
    float* supp_row = (float*)((char*)pSuppressed + y * nSuppressedStep);
    
    float magnitude = mag_row[x];
    float direction = dir_row[x];
    
    // 量化方向到4个主要方向
    float angle = direction;
    if (angle > 180) angle -= 180;
    
    int dx1, dy1, dx2, dy2;
    if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle <= 180)) {
        // 水平方向
        dx1 = 1; dy1 = 0; dx2 = -1; dy2 = 0;
    } else if (angle >= 22.5 && angle < 67.5) {
        // 对角线方向 /
        dx1 = 1; dy1 = -1; dx2 = -1; dy2 = 1;
    } else if (angle >= 67.5 && angle < 112.5) {
        // 垂直方向
        dx1 = 0; dy1 = 1; dx2 = 0; dy2 = -1;
    } else {
        // 对角线方向 \
        dx1 = -1; dy1 = -1; dx2 = 1; dy2 = 1;
    }
    
    // 检查相邻像素
    float mag1 = 0.0f, mag2 = 0.0f;
    
    int x1 = x + dx1, y1 = y + dy1;
    int x2 = x + dx2, y2 = y + dy2;
    
    if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
        const float* mag1_row = (const float*)((const char*)pGradMag + y1 * nMagStep);
        mag1 = mag1_row[x1];
    }
    
    if (x2 >= 0 && x2 < width && y2 >= 0 && y2 < height) {
        const float* mag2_row = (const float*)((const char*)pGradMag + y2 * nMagStep);
        mag2 = mag2_row[x2];
    }
    
    // 非最大值抑制
    if (magnitude >= mag1 && magnitude >= mag2) {
        supp_row[x] = magnitude;
    } else {
        supp_row[x] = 0.0f;
    }
}

// 第四步：双阈值检测和边缘连接
__global__ void cannyDoubleThreshold_kernel(const float* pSuppressed, int nSuppressedStep,
                                            Npp8u* pDst, int nDstStep,
                                            int width, int height,
                                            float nLowThreshold, float nHighThreshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    const float* supp_row = (const float*)((const char*)pSuppressed + y * nSuppressedStep);
    Npp8u* dst_row = (Npp8u*)((char*)pDst + y * nDstStep);
    
    float magnitude = supp_row[x];
    
    if (magnitude >= nHighThreshold) {
        dst_row[x] = 255;  // 强边缘
    } else if (magnitude >= nLowThreshold) {
        dst_row[x] = 128;  // 弱边缘
    } else {
        dst_row[x] = 0;    // 非边缘
    }
}

// 边缘连接（简化版）
__global__ void cannyEdgeTracing_kernel(Npp8u* pDst, int nDstStep, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    Npp8u* dst_row = (Npp8u*)((char*)pDst + y * nDstStep);
    
    if (dst_row[x] == 128) {  // 弱边缘
        // 检查8邻域是否有强边缘
        bool hasStrongNeighbor = false;
        
        for (int dy = -1; dy <= 1; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                if (dx == 0 && dy == 0) continue;
                
                int nx = x + dx;
                int ny = y + dy;
                
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    const Npp8u* neighbor_row = (const Npp8u*)((const char*)pDst + ny * nDstStep);
                    if (neighbor_row[nx] == 255) {
                        hasStrongNeighbor = true;
                        break;
                    }
                }
            }
            if (hasStrongNeighbor) break;
        }
        
        dst_row[x] = hasStrongNeighbor ? 255 : 0;
    }
}

extern "C" {

// 获取Canny边缘检测所需缓冲区大小
NppStatus nppiFilterCannyBorderGetBufferSize_8u_C1R_Ctx_cuda(NppiSize oSizeROI, int* hpBufferSize) {
    size_t imageSize = (size_t)oSizeROI.width * oSizeROI.height;
    
    // 需要多个临时缓冲区：
    // 1. 高斯滤波结果 (float)
    // 2. 梯度强度 (float)  
    // 3. 梯度方向 (float)
    // 4. 非最大值抑制结果 (float)
    
    size_t tempBufferSize = imageSize * sizeof(float) * 4;  // 4个float缓冲区
    size_t alignedSize = (tempBufferSize + 511) & ~511;     // 512字节对齐
    
    *hpBufferSize = (int)alignedSize;
    return NPP_SUCCESS;
}

// Canny边缘检测主函数
NppStatus nppiFilterCannyBorder_8u_C1R_Ctx_cuda(const Npp8u* pSrc, int nSrcStep, 
                                                NppiSize oSrcSizeROI, NppiPoint oSrcOffset,
                                                Npp8u* pDst, int nDstStep, NppiSize oDstSizeROI,
                                                NppiMaskSize eMaskSize, Npp32f nLowThreshold, 
                                                Npp32f nHighThreshold, NppiBorderType eBorderType,
                                                Npp8u* pDeviceBuffer, NppStreamContext nppStreamCtx) {
    
    int srcWidth = oSrcSizeROI.width;
    int srcHeight = oSrcSizeROI.height;
    int dstWidth = oDstSizeROI.width;
    int dstHeight = oDstSizeROI.height;
    
    // 设置临时缓冲区
    size_t imageSize = srcWidth * srcHeight;
    float* pBlurred = (float*)pDeviceBuffer;
    float* pGradMag = pBlurred + imageSize;
    float* pGradDir = pGradMag + imageSize;
    float* pSuppressed = pGradDir + imageSize;
    
    int floatStep = srcWidth * sizeof(float);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((srcWidth + blockSize.x - 1) / blockSize.x,
                  (srcHeight + blockSize.y - 1) / blockSize.y);
    
    // 第一步：高斯滤波
    cannyGaussianBlur_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, pBlurred, floatStep, srcWidth, srcHeight, eMaskSize, eBorderType);
    
    // 第二步：计算梯度
    cannySobelGradient_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pBlurred, floatStep, pGradMag, floatStep, pGradDir, floatStep, 
        srcWidth, srcHeight, eBorderType);
    
    // 第三步：非最大值抑制
    cannyNonMaxSuppression_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pGradMag, floatStep, pGradDir, floatStep, pSuppressed, floatStep, 
        srcWidth, srcHeight);
    
    // 调整网格大小用于输出
    dim3 dstGridSize((dstWidth + blockSize.x - 1) / blockSize.x,
                     (dstHeight + blockSize.y - 1) / blockSize.y);
    
    // 第四步：双阈值检测
    cannyDoubleThreshold_kernel<<<dstGridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSuppressed, floatStep, pDst, nDstStep, dstWidth, dstHeight, 
        nLowThreshold, nHighThreshold);
    
    // 第五步：边缘连接（多次迭代）
    for (int i = 0; i < 3; i++) {
        cannyEdgeTracing_kernel<<<dstGridSize, blockSize, 0, nppStreamCtx.hStream>>>(
            pDst, nDstStep, dstWidth, dstHeight);
    }
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

} // extern "C"