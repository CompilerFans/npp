#include "npp.h"
#include <hip/hip_runtime.h>


/**
 * CUDA kernels for NPP Box Filter with Border Functions
 */

// Device function to handle border pixel access
template<typename T>
__device__ T getBorderPixel(const T* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                           int x, int y, NppiBorderType eBorderType, T borderValue = 0) {
    switch (eBorderType) {
        case NPP_BORDER_REPLICATE:
            x = max(0, min(x, oSrcSizeROI.width - 1));
            y = max(0, min(y, oSrcSizeROI.height - 1));
            break;
        case NPP_BORDER_WRAP:
            x = (x + oSrcSizeROI.width) % oSrcSizeROI.width;
            y = (y + oSrcSizeROI.height) % oSrcSizeROI.height;
            break;
        case NPP_BORDER_MIRROR:
            if (x < 0) x = -x - 1;
            if (x >= oSrcSizeROI.width) x = 2 * oSrcSizeROI.width - x - 1;
            if (y < 0) y = -y - 1;
            if (y >= oSrcSizeROI.height) y = 2 * oSrcSizeROI.height - y - 1;
            break;
        case NPP_BORDER_CONSTANT:
            if (x < 0 || x >= oSrcSizeROI.width || y < 0 || y >= oSrcSizeROI.height) {
                return borderValue;
            }
            break;
        default:
            return borderValue;
    }
    
    const T* src_row = (const T*)((const char*)pSrc + y * nSrcStep);
    return src_row[x];
}

// Kernel for 8-bit unsigned single channel box filter with border
__global__ void nppiFilterBoxBorder_8u_C1R_kernel(const Npp8u* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                                  NppiPoint oSrcOffset, Npp8u* pDst, int nDstStep, NppiSize oDstSizeROI,
                                                  NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (dst_x < oDstSizeROI.width && dst_y < oDstSizeROI.height) {
        Npp8u* dst_row = (Npp8u*)((char*)pDst + dst_y * nDstStep);
        
        int sum = 0;
        int count = 0;
        
        // Apply box filter
        for (int ky = 0; ky < oMaskSize.height; ky++) {
            for (int kx = 0; kx < oMaskSize.width; kx++) {
                int src_x = dst_x + oSrcOffset.x + kx - oAnchor.x;
                int src_y = dst_y + oSrcOffset.y + ky - oAnchor.y;
                
                Npp8u pixel = getBorderPixel<Npp8u>(pSrc, nSrcStep, oSrcSizeROI, 
                                                    src_x, src_y, eBorderType, 0);
                sum += pixel;
                count++;
            }
        }
        
        dst_row[dst_x] = (count > 0) ? (Npp8u)(sum / count) : 0;
    }
}

// Kernel for 8-bit unsigned three channel box filter with border
__global__ void nppiFilterBoxBorder_8u_C3R_kernel(const Npp8u* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                                  NppiPoint oSrcOffset, Npp8u* pDst, int nDstStep, NppiSize oDstSizeROI,
                                                  NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (dst_x < oDstSizeROI.width && dst_y < oDstSizeROI.height) {
        Npp8u* dst_row = (Npp8u*)((char*)pDst + dst_y * nDstStep);
        
        int sum[3] = {0, 0, 0};
        int count = 0;
        
        // Apply box filter to each channel
        for (int ky = 0; ky < oMaskSize.height; ky++) {
            for (int kx = 0; kx < oMaskSize.width; kx++) {
                int src_x = dst_x + oSrcOffset.x + kx - oAnchor.x;
                int src_y = dst_y + oSrcOffset.y + ky - oAnchor.y;
                
                // Handle border for each channel
                for (int c = 0; c < 3; c++) {
                    if (src_x >= 0 && src_x < oSrcSizeROI.width && 
                        src_y >= 0 && src_y < oSrcSizeROI.height) {
                        const Npp8u* src_row = (const Npp8u*)((const char*)pSrc + src_y * nSrcStep);
                        sum[c] += src_row[src_x * 3 + c];
                    } else {
                        // Apply border handling per channel
                        switch (eBorderType) {
                            case NPP_BORDER_REPLICATE: {
                                int border_x = max(0, min(src_x, oSrcSizeROI.width - 1));
                                int border_y = max(0, min(src_y, oSrcSizeROI.height - 1));
                                const Npp8u* border_row = (const Npp8u*)((const char*)pSrc + border_y * nSrcStep);
                                sum[c] += border_row[border_x * 3 + c];
                                break;
                            }
                            case NPP_BORDER_CONSTANT:
                            default:
                                sum[c] += 0;  // Constant border value
                                break;
                        }
                    }
                }
                count++;
            }
        }
        
        // Store averaged results
        int dst_idx = dst_x * 3;
        for (int c = 0; c < 3; c++) {
            dst_row[dst_idx + c] = (count > 0) ? (Npp8u)(sum[c] / count) : 0;
        }
    }
}

// Kernel for 16-bit signed single channel box filter with border
__global__ void nppiFilterBoxBorder_16s_C1R_kernel(const Npp16s* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                                   NppiPoint oSrcOffset, Npp16s* pDst, int nDstStep, NppiSize oDstSizeROI,
                                                   NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (dst_x < oDstSizeROI.width && dst_y < oDstSizeROI.height) {
        Npp16s* dst_row = (Npp16s*)((char*)pDst + dst_y * nDstStep);
        
        int sum = 0;
        int count = 0;
        
        // Apply box filter
        for (int ky = 0; ky < oMaskSize.height; ky++) {
            for (int kx = 0; kx < oMaskSize.width; kx++) {
                int src_x = dst_x + oSrcOffset.x + kx - oAnchor.x;
                int src_y = dst_y + oSrcOffset.y + ky - oAnchor.y;
                
                Npp16s pixel = getBorderPixel<Npp16s>(pSrc, nSrcStep, oSrcSizeROI, 
                                                     src_x, src_y, eBorderType, 0);
                sum += pixel;
                count++;
            }
        }
        
        dst_row[dst_x] = (count > 0) ? (Npp16s)(sum / count) : 0;
    }
}

// Kernel for 32-bit float single channel box filter with border
__global__ void nppiFilterBoxBorder_32f_C1R_kernel(const Npp32f* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                                   NppiPoint oSrcOffset, Npp32f* pDst, int nDstStep, NppiSize oDstSizeROI,
                                                   NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (dst_x < oDstSizeROI.width && dst_y < oDstSizeROI.height) {
        Npp32f* dst_row = (Npp32f*)((char*)pDst + dst_y * nDstStep);
        
        float sum = 0.0f;
        int count = 0;
        
        // Apply box filter
        for (int ky = 0; ky < oMaskSize.height; ky++) {
            for (int kx = 0; kx < oMaskSize.width; kx++) {
                int src_x = dst_x + oSrcOffset.x + kx - oAnchor.x;
                int src_y = dst_y + oSrcOffset.y + ky - oAnchor.y;
                
                Npp32f pixel = getBorderPixel<Npp32f>(pSrc, nSrcStep, oSrcSizeROI, 
                                                      src_x, src_y, eBorderType, 0.0f);
                sum += pixel;
                count++;
            }
        }
        
        dst_row[dst_x] = (count > 0) ? (sum / count) : 0.0f;
    }
}

extern "C" {

// 8-bit unsigned single channel implementation
NppStatus nppiFilterBoxBorder_8u_C1R_Ctx_cuda(const Npp8u* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                              NppiPoint oSrcOffset, Npp8u* pDst, int nDstStep, NppiSize oDstSizeROI,
                                              NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType,
                                              NppStreamContext nppStreamCtx) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oDstSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oDstSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiFilterBoxBorder_8u_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, oSrcSizeROI, oSrcOffset, pDst, nDstStep, oDstSizeROI,
        oMaskSize, oAnchor, eBorderType);
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

// 8-bit unsigned three channel implementation
NppStatus nppiFilterBoxBorder_8u_C3R_Ctx_cuda(const Npp8u* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                              NppiPoint oSrcOffset, Npp8u* pDst, int nDstStep, NppiSize oDstSizeROI,
                                              NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType,
                                              NppStreamContext nppStreamCtx) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oDstSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oDstSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiFilterBoxBorder_8u_C3R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, oSrcSizeROI, oSrcOffset, pDst, nDstStep, oDstSizeROI,
        oMaskSize, oAnchor, eBorderType);
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

// 16-bit signed single channel implementation
NppStatus nppiFilterBoxBorder_16s_C1R_Ctx_cuda(const Npp16s* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                               NppiPoint oSrcOffset, Npp16s* pDst, int nDstStep, NppiSize oDstSizeROI,
                                               NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType,
                                               NppStreamContext nppStreamCtx) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oDstSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oDstSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiFilterBoxBorder_16s_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, oSrcSizeROI, oSrcOffset, pDst, nDstStep, oDstSizeROI,
        oMaskSize, oAnchor, eBorderType);
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

// 32-bit float single channel implementation
NppStatus nppiFilterBoxBorder_32f_C1R_Ctx_cuda(const Npp32f* pSrc, int nSrcStep, NppiSize oSrcSizeROI,
                                               NppiPoint oSrcOffset, Npp32f* pDst, int nDstStep, NppiSize oDstSizeROI,
                                               NppiSize oMaskSize, NppiPoint oAnchor, NppiBorderType eBorderType,
                                               NppStreamContext nppStreamCtx) {
    dim3 blockSize(16, 16);
    dim3 gridSize((oDstSizeROI.width + blockSize.x - 1) / blockSize.x,
                  (oDstSizeROI.height + blockSize.y - 1) / blockSize.y);
    
    nppiFilterBoxBorder_32f_C1R_kernel<<<gridSize, blockSize, 0, nppStreamCtx.hStream>>>(
        pSrc, nSrcStep, oSrcSizeROI, oSrcOffset, pDst, nDstStep, oDstSizeROI,
        oMaskSize, oAnchor, eBorderType);
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        return NPP_CUDA_KERNEL_EXECUTION_ERROR;
    }
    
    return NPP_SUCCESS;
}

} // extern "C"