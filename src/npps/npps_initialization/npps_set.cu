#include "npp.h"
#include <hip/hip_runtime.h>



// ==============================================================================
// GPU Kernels for Set Operations
// ==============================================================================


__global__ void nppsSet_8u_kernel_impl(Npp8u nValue, Npp8u *pDst, size_t nLength) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nLength) {
    pDst[idx] = nValue;
  }
}


__global__ void nppsSet_32f_kernel_impl(Npp32f nValue, Npp32f *pDst, size_t nLength) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nLength) {
    pDst[idx] = nValue;
  }
}


__global__ void nppsSet_32fc_kernel_impl(Npp32fc nValue, Npp32fc *pDst, size_t nLength) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < nLength) {
    pDst[idx].re = nValue.re;
    pDst[idx].im = nValue.im;
  }
}

// ==============================================================================
// Kernel Launch Functions
// ==============================================================================

extern "C" {

hipError_t nppsSet_8u_kernel(Npp8u nValue, Npp8u *pDst, size_t nLength, hipStream_t stream) {
  const int blockSize = 256;
  const int gridSize = (nLength + blockSize - 1) / blockSize;

  nppsSet_8u_kernel_impl<<<gridSize, blockSize, 0, stream>>>(nValue, pDst, nLength);

  return hipGetLastError();
}

hipError_t nppsSet_32f_kernel(Npp32f nValue, Npp32f *pDst, size_t nLength, hipStream_t stream) {
  const int blockSize = 256;
  const int gridSize = (nLength + blockSize - 1) / blockSize;

  nppsSet_32f_kernel_impl<<<gridSize, blockSize, 0, stream>>>(nValue, pDst, nLength);

  return hipGetLastError();
}

hipError_t nppsSet_32fc_kernel(Npp32fc nValue, Npp32fc *pDst, size_t nLength, hipStream_t stream) {
  const int blockSize = 256;
  const int gridSize = (nLength + blockSize - 1) / blockSize;

  nppsSet_32fc_kernel_impl<<<gridSize, blockSize, 0, stream>>>(nValue, pDst, nLength);

  return hipGetLastError();
}

} // extern "C"
